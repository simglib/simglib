#include "hip/hip_runtime.h"
/// \file cufft2d.cpp
/// \brief cufft2d definitions
/// \author Sylvain Prigent
/// \version 0.1
/// \date 2020

#include <iostream>
#include <hipfft/hipfft.h>

#include <smanipulate>

__global__
void convolve_fft(unsigned int Nfft, float scale, hipfftComplex* image1FFT, hipfftComplex* image2FFT, hipfftComplex* outputFFT)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < Nfft)
    {
        outputFFT[i].x = (image1FFT[i].x * image2FFT[i].x - image1FFT[i].y * image2FFT[i].y)*scale;
        outputFFT[i].y = (image1FFT[i].y * image2FFT[i].x + image1FFT[i].x * image2FFT[i].y)*scale;
    }
}

namespace SImg
{

    void convolve2d(float *image1, float* image2, unsigned int sx, unsigned int sy, float*output)
    {

        // shift image 2
        float *image2Shift = new float[sx * sy];
        shift2D(image2, image2Shift, sx, sy, int(-float(sx) / 2.0), int(-float(sy) / 2.0));

        int N = sx * sy;
        int Nfft = sx*(sy/2+1);
        float scale =  1.0 / float(sx*sy);
        float* d_image1;
        float* d_image2;
        float* d_output;
        hipfftComplex *d_image1FFT;
        hipfftComplex *d_image2FFT;
        hipfftComplex *d_outputFFT;

        // alloc
        hipMalloc(&d_image1, N * sizeof(float)); 
        hipMalloc(&d_image2, N * sizeof(float)); 
        hipMalloc(&d_output, N * sizeof(float)); 
        hipMalloc((void**)&d_image1FFT, sizeof(hipfftComplex)*Nfft); 
        hipMalloc((void**)&d_image2FFT, sizeof(hipfftComplex)*Nfft); 
        hipMalloc((void**)&d_outputFFT, sizeof(hipfftComplex)*Nfft); 

        // copy inputs
        hipMemcpy(d_image1, image1, N*sizeof(float), hipMemcpyHostToDevice); 
        hipMemcpy(d_image2, image2Shift, N*sizeof(float), hipMemcpyHostToDevice); 

        // convolution
        hipfftHandle planR2C;
        hipfftPlan2d(&planR2C, sx, sy, HIPFFT_R2C);

        hipfftExecR2C(planR2C, (hipfftReal*)d_image1, (hipfftComplex*)d_image1FFT);
        hipfftExecR2C(planR2C, (hipfftReal*)d_image2, (hipfftComplex*)d_image2FFT);

        int blockSize1d = 256;
        int numBlocks1d = (Nfft + blockSize1d - 1) / blockSize1d;
        convolve_fft<<<numBlocks1d,blockSize1d>>>(Nfft, scale, d_image1FFT, d_image2FFT, d_outputFFT);

        hipfftHandle planC2R;
        hipfftPlan2d(&planC2R, sx, sy, HIPFFT_C2R);
        hipfftExecC2R(planC2R, (hipfftComplex*)d_outputFFT, (hipfftReal*)d_output);
        hipMemcpy(output, d_output, N*sizeof(float), hipMemcpyDeviceToHost);

        hipFree(d_image1);
        hipFree(d_image2);
        hipFree(d_output);
        hipFree(d_image1FFT);
        hipFree(d_image2FFT);
        hipFree(d_outputFFT);

    }

    void cufft2d(float *image, unsigned int sx, unsigned int sy, float *output)
    {
        int bs = sx * sy;
        float *cuimage;
        float *cuoutput;
        hipfftComplex *cuimageFFT;

        // allocate input to GPU
        std::cout << "copy image to cuda" << std::endl;
        hipMalloc(&cuimage, bs * sizeof(float));   
        hipMalloc(&cuoutput, bs * sizeof(float)); 
        hipMemcpy(cuimage, image, bs*sizeof(float), hipMemcpyHostToDevice); 
        hipMalloc((void**)&cuimageFFT, sizeof(hipfftComplex)*sx*(sy/2+1));
        
        hipfftHandle pF, pI;
        hipfftPlan2d(&pF, sx, sy, HIPFFT_R2C);
        hipfftPlan2d(&pI, sx, sy, HIPFFT_C2R);

        hipfftExecR2C(pF, (hipfftReal*)cuimage, (hipfftComplex*)cuimageFFT);
        hipfftExecC2R(pI, (hipfftComplex*)cuimageFFT, (hipfftReal*)cuoutput);

        hipfftDestroy(pF);
        hipfftDestroy(pI);

        // copy output to cpu
        std::cout << "copy memory to output" << std::endl;
        hipMemcpy(output, cuoutput, bs*sizeof(float), hipMemcpyDeviceToHost);
    }
}