#include "hip/hip_runtime.h"
/// \file srichardsonlucy.cu
/// \brief srichardsonlucy definitions
/// \author Sylvain Prigent
/// \version 0.1
/// \date 2020


#include "srichardsonlucy.h"
#include <smanipulate>
#include <hipfft/hipfft.h>

#include <iostream>

__global__
void rl_init_buffer_out(unsigned int N, float* buffer_out)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N)
    {
        buffer_out[i] = 0.5;
    }
}

__global__
void rl_mirror_psf(unsigned int sx, unsigned int sy, float* psf_mirror, float* psf)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < 0 || x >= sx || y < 0 || y >= sy)
    {
        return;
    }
    psf_mirror[sy * x + y] = psf[sy * x + (sy - 1 - y)];
}

__global__
void rl_convolve(unsigned int n_fft, float scale, hipfftComplex *image1, hipfftComplex *image2, hipfftComplex *image_out)
{
    int p = blockIdx.x * blockDim.x + threadIdx.x;
    if (p < n_fft)
    {
        image_out[p].x = (image1[p].x * image2[p].x - image1[p].y * image2[p].y) * scale;
        image_out[p].y = (image1[p].y * image2[p].x + image1[p].x * image2[p].y) * scale;
    }
}

__global__
void rl_normalize_tmp(unsigned int n, float* tmp, float* buffer_in)
{
    int p = blockIdx.x * blockDim.x + threadIdx.x;
    if (p < n)
    {
        if (tmp[p] > 1e-9)
        {
            tmp[p] = buffer_in[p] / tmp[p];
        }
        else
        {
            tmp[p] = 0;
        }
    }
}

__global__
void rl_update_iter(unsigned int n, float* buffer_out, float* tmp)
{
    int p = blockIdx.x * blockDim.x + threadIdx.x;
    if (p < n){
        for (unsigned int p = 0; p < n; p++)
        {
            buffer_out[p] *= tmp[p];
        }
    }
}


namespace SImg{

    void cuda_richardsonlucy_2d(float* buffer_in, float* psf, float* buffer_out, unsigned int sx, unsigned int sy, unsigned int niter)
    {
        // memory
        unsigned int n = sx * sy;
        unsigned int n_fft = sx * (sy / 2 + 1);
        float scale = 1.0 / float(n_fft);

        hipfftComplex *fft_in;
        hipfftComplex *fft_out;
        hipfftComplex *fft_psf;
        hipfftComplex *fft_psf_mirror;
        hipfftComplex *fft_tmp;
        float *psf_mirror;
        float *tmp;
        float* cu_buffer_in;
        float* cu_psf;
        float* cu_buffer_out;

        hipMalloc((void**)&fft_in, sizeof(hipfftComplex)*n_fft);
        hipMalloc((void**)&fft_out, sizeof(hipfftComplex)*n_fft);
        hipMalloc((void**)&fft_psf, sizeof(hipfftComplex)*n_fft);
        hipMalloc((void**)&fft_psf_mirror, sizeof(hipfftComplex)*n_fft);
        hipMalloc((void**)&fft_tmp, sizeof(hipfftComplex)*n_fft);
        hipMalloc ( &psf_mirror, n*sizeof(float));
        hipMalloc ( &tmp, n*sizeof(float));
        hipMalloc ( &cu_buffer_in, n*sizeof(float));
        hipMalloc ( &cu_psf, n*sizeof(float));
        hipMalloc ( &cu_buffer_out, n*sizeof(float));

        hipMemcpy(cu_buffer_in, buffer_in, n*sizeof(float), hipMemcpyHostToDevice);

        hipfftHandle Planfft;
        hipfftPlan2d(&Planfft, sx, sy, HIPFFT_R2C);
        hipfftHandle Planifft;
        hipfftPlan2d(&Planifft, sx, sy, HIPFFT_C2R);

        int blockSize1d = 256;
        int numBlocks1d = (n + blockSize1d - 1) / blockSize1d;
        int numBlocks1dfft = (n_fft + blockSize1d - 1) / blockSize1d;
        dim3 blockSize2d(16, 16);
        dim3 gridSize2d = dim3((sx + 16 - 1) / 16, (sy + 16 - 1) / 16);

        // initialization
        std::cout << "initialization" << std::endl;
        hipfftExecR2C(Planfft, (hipfftReal*)cu_buffer_in, (hipfftComplex*)fft_in);

        rl_init_buffer_out<<<numBlocks1d, blockSize1d>>>(n, cu_buffer_out);

        float *psf_shifted = new float[sx * sy];
        shift2D(psf, psf_shifted, sx, sy, int(-float(sx) / 2.0), int(-float(sy) / 2.0));
        float* cu_psf_shifted;
        hipMalloc ( &cu_psf_shifted, n*sizeof(float));
        hipMemcpy(cu_psf_shifted, psf_shifted, n*sizeof(float), hipMemcpyHostToDevice);
        hipfftExecR2C(Planfft, (hipfftReal*)cu_psf_shifted, (hipfftComplex*)fft_psf);
        hipFree(cu_psf_shifted);
        delete[] psf_shifted;

        // flip psf
        std::cout << "flip psf" << std::endl;
        rl_mirror_psf<<<gridSize2d,blockSize2d>>>(sx, sy, psf_mirror, cu_psf);
        hipfftExecR2C(Planfft, (hipfftReal*)psf_mirror, (hipfftComplex*)fft_psf_mirror);

        hipFree(psf_mirror);

        unsigned int iter = 0;
        std::cout << "iterations" << std::endl;
        hipDeviceSynchronize();
        while (iter < niter)
        {
            iter++;
            // tmp = convolve(buffer_out, psf)
            hipfftExecR2C(Planfft, (hipfftReal*)buffer_out, (hipfftComplex*)fft_out);
            rl_convolve<<<numBlocks1dfft,blockSize1d>>>(n_fft, scale, fft_out, fft_psf, fft_tmp);
            hipfftExecC2R(Planifft, (hipfftComplex*)fft_tmp, (hipfftReal*)tmp);    

            // tmp = buffer_in / tmp
            rl_normalize_tmp<<<numBlocks1d, blockSize1d>>>(n, tmp, buffer_in);

            // im_deconv *= convolve(tmp, psf_mirror)
            hipfftExecR2C(Planfft, (hipfftReal*)tmp, (hipfftComplex*)fft_tmp);

            rl_convolve<<<numBlocks1dfft,blockSize1d>>>(n_fft, scale, fft_tmp, fft_psf, fft_tmp);
            hipfftExecC2R(Planifft, (hipfftComplex*)fft_tmp, (hipfftReal*)tmp); 

            rl_update_iter<<<numBlocks1d, blockSize1d>>>(n, buffer_out, tmp);

        }
        std::cout << "copy and free" << std::endl;
        hipDeviceSynchronize();
        // free output
        hipfftDestroy(Planfft);
        hipfftDestroy(Planifft);

        hipMemcpy(buffer_out, cu_buffer_out, n*sizeof(float), hipMemcpyDeviceToHost);

        hipFree(cu_buffer_out);
        hipFree(cu_buffer_in);

        hipFree(fft_in);
        hipFree(fft_out);
        hipFree(fft_psf);
        hipFree(fft_psf_mirror);
        hipFree(fft_tmp);
        hipFree(tmp);
    }

    void cuda_richardson_lucy_3d(float* buffer_in, float* psf, float* buffer_out, unsigned int sx, unsigned int sy, unsigned int sz, unsigned int niter)
    {

    }
    
}