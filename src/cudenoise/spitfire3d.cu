#include "hip/hip_runtime.h"
/// \file spitfire2d.cpp
/// \brief spitfire2d definitions
/// \author Sylvain Prigent
/// \version 0.1
/// \date 2020


#include "spitfire2d.h"
#include <score/SMath.h>
#include <scli>

__global__
void init_3d_buffers_hv(unsigned int N, float* cu_denoised_image, float* cu_noisy_image, float* dual_images0, 
                     float* dual_images1, float* dual_images2, float* dual_images3,
                     float* dual_images4, float* dual_images5, float* dual_images6)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N)
    {
        cu_denoised_image[i] = cu_noisy_image[i];
        dual_images0[i] = 0.0;
        dual_images1[i] = 0.0;
        dual_images2[i] = 0.0;
        dual_images3[i] = 0.0;
        dual_images4[i] = 0.0;
        dual_images5[i] = 0.0;
        dual_images6[i] = 0.0;
    }
}

__global__
void init_3d_buffers_sv(unsigned int N, float* cu_denoised_image, float* cu_noisy_image, float* dual_images0, 
                     float* dual_images1, float* dual_images2, float* dual_images3)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N)
    {
        cu_denoised_image[i] = cu_noisy_image[i];
        dual_images0[i] = 0.0;
        dual_images1[i] = 0.0;
        dual_images2[i] = 0.0;
        dual_images3[i] = 0.0;
    }
}

__global__
void copy_buffer(float* in_buffer, unsigned int n, float *out_buffer)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
    {
        out_buffer[i] = in_buffer[i];
    }
}

__global__
void sv_3d_primal(unsigned int sx, unsigned int sy, unsigned int sz, float primal_step, float primal_weight, 
                  float primal_weight_comp, float *denoised_image, float *noisy_image,
                  float* dual_images0, float* dual_images1, float* dual_images2, float* dual_images3)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int y = blockIdx.z * blockDim.z + threadIdx.z;
    if (x < 1 || x >= sx-1 || y < 1 || y >= sy-1 || z < 1 || z >= sz-1)
    {
        return;
    }

    unsigned int p = z + sz * (y + sy * x);
    unsigned int pxm = p - sz * sy;
    unsigned int pym = p - sz;
    unsigned int pzm = p - 1;

    float tmp = denoised_image[p] - primal_step * (denoised_image[p] - noisy_image[p]);

    float dx_adj = dual_images0[pxm] - dual_images0[p];
    float dy_adj = dual_images1[pym] - dual_images1[p];
    float dz_adj = delta * (dual_images2[pzm] - dual_images2[p]);

    tmp -= (primal_weight * (dx_adj + dy_adj + dz_adj) + primal_weight_comp * dual_images3[p]);

    if (tmp > 1.0)
    {
        denoised_image[p] = 1.0;
    }
    else if (tmp < 0.0)
    {
        denoised_image[p] = 0.0;
    }
    else
    {
        denoised_image[p] = tmp;
    }   

}

__global__
void hv_3d_primal(unsigned int sx, unsigned int sy, unsigned int sz, float primal_step, float primal_weight, float primal_weight_comp, float sqrt2, float *denoised_image, float *noisy_image,
                  float* dual_images0, float* dual_images1, float* dual_images2, float* dual_images3,
                  float* dual_images4, float* dual_images5, float* dual_images6)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;
    if (x < 1 || x >= sx-1 || y < 1 || y >= sy-1|| z < 1 || z >= sz-1)
    {
        return;
    }
    unsigned int p = z + sz * (y + sy * x);
    unsigned int pxm = p - sz * sy;
    unsigned int pym = p - sz;
    unsigned int pzm = p - 1;
    unsigned int pxp = p + sz * sy;
    unsigned int pyp = p + sz;
    unsigned int pzp = p + 1;

    float tmp = denoised_image[p] - primal_step * (denoised_image[p] - noisy_image[p]);

    float dxx_adj = dual_images0[pxm] - 2 * dual_images0[p] + dual_images0[pxp];
    float dyy_adj = dual_images1[pym] - 2 * dual_images1[p] + dual_images1[pyp];
    float dzz_adj = (delta * delta) * (dual_images2[pzm] - 2 * dual_images2[p] + dual_images2[pzp]);

    // Other terms
    float dxy_adj = dual_images3[p] - dual_images3[pxm] - dual_images3[pym] + dual_images3[z + sz * (y - 1 + sy * (x - 1))];
    float dyz_adj = delta * (dual_images4[p] - dual_images4[pym] - dual_images4[pzm] + dual_images4[z - 1 + sz * (y - 1 + sy * x)]);
    float dzx_adj = delta * (dual_images5[p] - dual_images5[pzm] - dual_images5[pxm] + dual_images5[z - 1 + sz * (y + sy * (x - 1))]);

    tmp -= (primal_weight * (dxx_adj + dyy_adj + dzz_adj + sqrt2 * (dxy_adj + dyz_adj + dzx_adj)) + primal_weight_comp * dual_images6[p]);

    if (tmp > 1.0)
    {
        denoised_image[p] = 1.0;
    }
    else if (tmp < 0.0)
    {
        denoised_image[p] = 0.0;
    }
    else
    {
        denoised_image[p] = tmp;
    }
}

__global__
void dual_3d_auxiliary(unsigned int N , float* auxiliary_image, float* denoised_image)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N)
    {
        auxiliary_image[i] = 2 * denoised_image[i] - auxiliary_image[i];
    }
}

__global__
void sv_3d_dual(unsigned int sx, unsigned int sy, float dual_weight, float dual_weight_comp, float*auxiliary_image, float* dual_images0, float* dual_images1,
                float* dual_images2, float* dual_images3)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;
    if (x < 1 || x >= sx-1 || y < 1 || y >= sy-1 || z < 1 || z >= sz-1)
    {
        return;
    }

    unsigned int p = z + sz * (y + sy * x);
    unsigned int pxp = p + sz * sy;
    unsigned int pyp = p + sz;
    unsigned int pzp = p + 1;

    dual_images0[p] += dual_weight * (auxiliary_image[pxp] - auxiliary_image[p]);
    dual_images1[p] += dual_weight * (auxiliary_image[pyp] - auxiliary_image[p]);
    dual_images2[p] += dual_weight * (delta * (auxiliary_image[pzp] - auxiliary_image[p]));
    dual_images3[p] += dual_weight_comp * auxiliary_image[p];
}

__global__
void hv_3d_dual(unsigned int sx, unsigned int sy, float dual_weight, float dual_weight_comp, float sqrt2, float*auxiliary_image, float* dual_images0, float* dual_images1,
                float* dual_images2, float* dual_images3, float* dual_images4, float* dual_images5, float* dual_images6)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;
    if (x < 1 || x >= sx-1 || y < 1 || y >= sy-1 || z < 1 || z >= sz-1)
    {
        return;
    }
    unsigned int p = z + sz * (y + sy * x);  
    unsigned int pxm = p - sz*sy;
    unsigned int pym = p - sz;  
    unsigned int pzm = p - 1;
    unsigned int pxp = p + sz*sy;
    unsigned int pyp = p + sz;
    unsigned int pzp = p + 1;  

    dual_images0[p] += dual_weight * (auxiliary_image[pxp] - 2 * auxiliary_image[p] + auxiliary_image[pxm]);
    dual_images1[p] += dual_weight * (auxiliary_image[pyp] - 2 * auxiliary_image[p] + auxiliary_image[pym]);
    dual_images2[p] += dual_weight * ((delta * delta) * (auxiliary_image[pzp] - 2 * auxiliary_image[p] + auxiliary_image[pzm]));
    dual_images3[p] += sqrt2 * dual_weight * (auxiliary_image[z + sz * (y + 1 + sy * (x + 1))] - auxiliary_image[pxp] - auxiliary_image[pyp] + auxiliary_image[p]);
    dual_images4[p] += sqrt2 * dual_weight * (delta * (auxiliary_image[z + 1 + sz * (y + 1 + sy * x)] - auxiliary_image[pyp] - auxiliary_image[pzp] + auxiliary_image[p]));
    dual_images5[p] += sqrt2 * dual_weight * (delta * (auxiliary_image[z + 1 + sz * (y + sy * (x + 1))] - auxiliary_image[pxp] - auxiliary_image[pzp] + auxiliary_image[p]));
    dual_images6[p] += dual_weight_comp * auxiliary_image[p];
}

__global__
void hv_dual_3d_normalize(unsigned int N, float inv_reg, float* dual_images0, float* dual_images1, float* dual_images2, float* dual_images3,
                          float* dual_images4, float* dual_images5, float* dual_images6)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N)
    {
        float tmp = inv_reg * sqrt( dual_images0[i]*dual_images0[i] + dual_images1[i]*dual_images1[i] + dual_images2[i]*dual_images2[i] + dual_images3[i]*dual_images3[i]
                                   + dual_images4[i] * dual_images4[i] + dual_images4[i] * dual_images4[i] + dual_images6[i] * dual_images6[i]);
        if (tmp > 1.0)
        {
            float inv_tmp = 1.0/tmp;
            dual_images0[i] *= inv_tmp;
            dual_images1[i] *= inv_tmp;
            dual_images2[i] *= inv_tmp;
            dual_images3[i] *= inv_tmp;
            dual_images4[i] *= inv_tmp;
            dual_images5[i] *= inv_tmp;
            dual_images6[i] *= inv_tmp;
        }
    }
}

__global__
void sv_dual_3d_normalize(unsigned int N, float inv_reg, float* dual_images0, float* dual_images1, float* dual_images2, float* dual_images3)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N)
    {
        float tmp = inv_reg * sqrt( dual_images0[i]*dual_images0[i] + dual_images1[i]*dual_images1[i] + dual_images2[i]*dual_images2[i] + dual_images3[i]*dual_images3[i]);
        if (tmp > 1.0)
        {
            float inv_tmp = 1.0/tmp;
            dual_images0[i] *= inv_tmp;
            dual_images1[i] *= inv_tmp;
            dual_images2[i] *= inv_tmp;
            dual_images3[i] *= inv_tmp;
        }
    }
}

namespace SImg{

    void cuda_spitfire3d_denoise_sv(float* noisy_image, unsigned int sx, unsigned int sy, unsigned int sz, float* denoised_image, const float& regularization, const float& weighting, const unsigned int& niter, bool verbose, SObservable* observable)
    {
        unsigned int N = sx*sy*sz;
    
        // Splitting parameters
        float dual_step = SMath::max(0.01, SMath::min(0.1, regularization));
        float primal_step = 0.99
                / (0.5
                   + (8 * pow(weighting, 2.)
                      + pow(1 - weighting, 2.)) * dual_step);
        float primal_weight = primal_step * weighting;
        float primal_weight_comp = primal_step * (1 - weighting);
        float dual_weight = dual_step * weighting;
        float dual_weight_comp = dual_step * (1 - weighting);
    
        // Initializations
        float* dual_images0;
        float* dual_images1;
        float* dual_images2;
        float* dual_images3;
        float* auxiliary_image;
        float* cu_denoised_image;
        float* cu_noisy_image;

        hipMalloc ( &dual_images0, N*sizeof(float));
        hipMalloc ( &dual_images1, N*sizeof(float));
        hipMalloc ( &dual_images2, N*sizeof(float));
        hipMalloc ( &dual_images3, N*sizeof(float));
        hipMalloc ( &auxiliary_image, N*sizeof(float));
        hipMalloc ( &cu_denoised_image, N*sizeof(float));
        hipMalloc ( &cu_noisy_image, N*sizeof(float));
        hipMemcpy(cu_noisy_image, noisy_image, N*sizeof(float), hipMemcpyHostToDevice); 

        //STimer timer;
        //timer.setObserver(new SObserverConsole());
        //timer.tic();

        // cida threads blocs
        int blockSize1d = 256;
        int numBlocks1d = (N + blockSize1d - 1) / blockSize1d;
        dim3 blockSize3d(16, 16, 16);
        dim3 gridSize3d = dim3((sx + 16 - 1) / 16, (sy + 16 - 1) / 16, (sz + 16 - 1) / 16);

        // init in cuda
        init_3d_buffers_sv<<<numBlocks1d, blockSize1d>>>(N, cu_denoised_image, cu_noisy_image, dual_images0, dual_images1, dual_images2, dual_images3);

        // Denoising process
        float inv_reg = 1.0 / regularization;
        for (int iter = 0; iter < niter; iter++) {
    
            // Primal optimization
            copy_buffer<<<numBlocks1d, blockSize1d>>>(cu_denoised_image, N, auxiliary_image);

            sv_3d_primal<<<blockSize3d,gridSize3d>>>(sx, sy, sz, primal_step, primal_weight, primal_weight_comp, 
                                                     cu_denoised_image, cu_noisy_image, dual_images0, dual_images1, 
                                                     dual_images2, dual_images3);
        
            // Stopping criterion
            if (verbose){
                int iter_n = niter / 10;
                if (iter_n < 1) iter_n = 1;
                if (iter % iter_n == 0){
                    observable->notifyProgress(100*(float(iter)/float(niter)));
                }
            }
    
            // Dual optimization
            dual_3d_auxiliary<<<numBlocks1d, blockSize1d>>>(N, auxiliary_image, cu_denoised_image);

            // dual    
            sv_3d_dual<<<blockSize3d,gridSize3d>>>(sx, sy, dual_weight, dual_weight_comp, 
                                                   auxiliary_image, dual_images0, 
                                                   dual_images1, dual_images2, dual_images3);

    
            // normalize
            sv_dual_3d_normalize<<<numBlocks1d, blockSize1d>>>(N, inv_reg, dual_images0, dual_images1, dual_images2, dual_images3);
                                                   
        } // endfor (int iter = 0; iter < nb_iters_max; iter++)
        hipDeviceSynchronize();
        //timer.toc();
        hipFree(dual_images0);
        hipFree(dual_images1);
        hipFree(dual_images2);
        hipFree(dual_images3);
        hipFree(auxiliary_image);
        
        hipMemcpy(denoised_image, cu_denoised_image, N*sizeof(float), hipMemcpyDeviceToHost);   
        hipFree(cu_denoised_image); 
        hipFree(cu_noisy_image);       
        
        if (verbose){
            observable->notifyProgress(100);
        }
    }

    void cuda_spitfire3d_denoise_hv(float* noisy_image, unsigned int sx, unsigned int sy, unsigned int sz, float* denoised_image, const float& regularization, const float& weighting, const unsigned int& niter, bool verbose, SObservable* observable)
    {
        unsigned int N = sx*sy*sz;
        float sqrt2 = sqrt(2.);
    
        // Splitting parameters
        float dual_step = SMath::max(0.001, SMath::min(0.01, regularization));
        float primal_step = 0.99 / (0.5 + (64 * pow(weighting, 2.) + pow(1 - weighting, 2.)) * dual_step);
        float primal_weight = primal_step * weighting;
        float primal_weight_comp = primal_step * (1 - weighting);
        float dual_weight = dual_step * weighting;
        float dual_weight_comp = dual_step * (1 - weighting);
    
        // Initializations
        float* dual_images0;
        float* dual_images1;
        float* dual_images2;
        float* dual_images3;
        float* dual_images4;
        float* dual_images5;
        float* dual_images6;
        float* auxiliary_image;
        float* cu_denoised_image;
        float* cu_noisy_image;

        hipMalloc ( &dual_images0, N*sizeof(float));
        hipMalloc ( &dual_images1, N*sizeof(float));
        hipMalloc ( &dual_images2, N*sizeof(float));
        hipMalloc ( &dual_images3, N*sizeof(float));
        hipMalloc ( &dual_images4, N*sizeof(float));
        hipMalloc ( &dual_images5, N*sizeof(float));
        hipMalloc ( &dual_images6, N*sizeof(float));
        hipMalloc ( &auxiliary_image, N*sizeof(float));
        hipMalloc ( &cu_denoised_image, N*sizeof(float));
        hipMalloc ( &cu_noisy_image, N*sizeof(float));
        hipMemcpy(cu_noisy_image, noisy_image, N*sizeof(float), hipMemcpyHostToDevice); 
    
        //STimer timer;
        //timer.setObserver(new SObserverConsole());
        //timer.tic();

        // cuda threads blocs
        int blockSize1d = 256;
        int numBlocks1d = (N + blockSize1d - 1) / blockSize1d;
        dim3 blockSize3d(16, 16, 16);
        dim3 gridSize3d = dim3((sx + 16 - 1) / 16, (sy + 16 - 1) / 16, (sz + 16 - 1) / 16);

        // init in cuda
        init_3d_buffers_hv<<<numBlocks1d, blockSize1d>>>(N, cu_denoised_image, cu_noisy_image, dual_images0, dual_images1, dual_images2, dual_images3, dual_images4, dual_images5, dual_images6);
        
        // Deconvolution process
        float inv_reg = 1.0 / regularization;
        for (int iter = 0; iter < niter; ++iter) {

            // Primal optimization
            copy_buffer<<<numBlocks1d, blockSize1d>>>(cu_denoised_image, N, auxiliary_image);
    
            hv_3d_primal<<<gridSize3d, blockSize3d>>>(sx, sy, primal_step, primal_weight, primal_weight_comp, sqrt2, 
                                                     cu_denoised_image, cu_noisy_image, dual_images0, dual_images1, 
                                                     dual_images2, dual_images3, dual_images4, dual_images5, dual_images6);
    
            // Stopping criterion
            if (verbose){
                int iter_n = niter / 10;
                if (iter_n < 1) iter_n = 1;
                if (iter % iter_n == 0){
                    observable->notifyProgress(100*(float(iter)/float(niter)));
                }
            }
    
            // Dual optimization
            // dual_auxilary
            dual_3d_auxiliary<<<numBlocks1d, blockSize1d>>>(N, auxiliary_image, cu_denoised_image);
    
            // dual    
            hv_3d_dual<<<gridSize2d, blockSize2d>>>(sx, sy, sy, dual_weight, dual_weight_comp, sqrt2, 
                                                   auxiliary_image, dual_images0, 
                                                   dual_images1, dual_images2, dual_images3,
                                                   dual_images4, dual_images5, dual_images6);
    
            //normlization    
            hv_dual_3d_normalize<<<numBlocks1d, blockSize1d>>>(N, inv_reg, dual_images0, dual_images1, 
                                                            dual_images2, dual_images3, dual_images4, 
                                                            dual_images5, dual_images6);    
        } // endfor (int iter = 0; iter < nb_iters_max; iter++)
        hipDeviceSynchronize();
        //timer.toc();
        hipFree(dual_images0);
        hipFree(dual_images1);
        hipFree(dual_images2);
        hipFree(dual_images3);
        hipFree(dual_images4);
        hipFree(dual_images5);
        hipFree(dual_images6);
        hipFree(auxiliary_image);
        
        hipMemcpy(denoised_image, cu_denoised_image, N*sizeof(float), hipMemcpyDeviceToHost);   
        hipFree(cu_denoised_image); 
        hipFree(cu_noisy_image);       
        
        if (verbose){
            observable->notifyProgress(100);
        }
    }

    void cuda_spitfire3d_denoise(float *blurry_image, unsigned int sx, unsigned int sy, unsigned int sz, float *psf, float *deconv_image, const float &regularization, const float &weighting, const unsigned int &niter, const std::string &method, bool verbose, SObservable *observable)
    {
        // normalize the input image
        unsigned int bs = sx * sy * sz;
        float imin = blurry_image[0];
        float imax = blurry_image[0];
        for (unsigned int i = 1; i < bs; ++i)
        {
            float val = blurry_image[i];
            if (val > imax)
            {
                imax = val;
            }
            if (val < imin)
            {
                imin = val;
            }
        }

        float *blurry_image_norm = new float[sx * sy * sz];
        normL2(blurry_image, sx, sy, sz, 1, 1, blurry_image_norm);

        // run denoising
        if (method == "SV")
        {
            cuda_spitfire3d_denoise_sv(blurry_image_norm, sx, sy, psf, deconv_image, regularization, weighting, niter, verbose, observable);
        }
        else if (method == "HV")
        {
            cuda_spitfire3d_denoise_hv(blurry_image_norm, sx, sy, psf, deconv_image, regularization, weighting, niter, verbose, observable);
        }
        else
        {
            throw SException("spitfire3d: method must be SV or HV");
        }

        // normalize back intensities
        float omin = deconv_image[0];
        float omax = deconv_image[0];
        for (unsigned int i = 1; i < bs; ++i)
        {
            float val = deconv_image[i];
            if (val > omax)
            {
                omax = val;
            }
            if (val < omin)
            {
                omin = val;
            }
        }

#pragma omp parallel for
        for (unsigned int i = 0; i < sx * sy * sz; ++i)
        {
           deconv_image[i] = (deconv_image[i] - omin)/(omax-omin);
           deconv_image[i] = deconv_image[i] * (imax - imin) + imin;
        }

        delete[] blurry_image_norm;
    }
}