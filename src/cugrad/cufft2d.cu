/// \file cufft2d.cpp
/// \brief cufft2d definitions
/// \author Sylvain Prigent
/// \version 0.1
/// \date 2020


#include <hip/hip_runtime.h>
#include <iostream>
#include <hipfft/hipfft.h>

namespace SImg
{
    void cufft2d(float *image, unsigned int sx, unsigned int sy, float *output)
    {
        int bs = sx * sy;
        float *cuimage;
        float *cuoutput;
        hipfftComplex *cuimageFFT;

        // allocate input to GPU
        std::cout << "copy image to cuda" << std::endl;
        hipMalloc(&cuimage, bs * sizeof(float));   
        hipMalloc(&cuoutput, bs * sizeof(float)); 
        hipMemcpy(cuimage, image, bs*sizeof(float), hipMemcpyHostToDevice); 
        hipMalloc((void**)&cuimageFFT, sizeof(hipfftComplex)*sx*(sy/2+1));
        
        hipfftHandle pF, pI;
        hipfftPlan2d(&pF, sx, sy, HIPFFT_R2C);
        hipfftPlan2d(&pI, sx, sy, HIPFFT_C2R);

        hipfftExecR2C(pF, (hipfftReal*)cuimage, (hipfftComplex*)cuimageFFT);
        hipfftExecC2R(pI, (hipfftComplex*)cuimageFFT, (hipfftReal*)cuoutput);

        // copy output to cpu
        std::cout << "copy memory to output" << std::endl;
        hipMemcpy(output, cuoutput, bs*sizeof(float), hipMemcpyDeviceToHost);
    }
}